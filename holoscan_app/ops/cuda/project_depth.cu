
#include <hip/hip_runtime.h>

// project_depth.cu — reference signature
extern "C" __global__
void project_overlay(
    const float* depth, int Hd, int Wd,
    unsigned char* rgb, int Hr, int Wr,
    const float* Kd, const float* Kr,
    const float* T,
    unsigned int* zbuf, int splat_radius,
    float z_min, float z_max, float alpha);
